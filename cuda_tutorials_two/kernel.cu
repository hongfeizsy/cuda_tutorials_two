#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define get_idx() (threadIdx.x)

__global__ void sum(float *x) {
	int idx = get_idx();
	x[idx] += 1;
}

int main() {
	int N = 32;
	int nbytes = N * sizeof(float);

	float *dx = NULL, *hx = NULL;

	/* allocate GPU memory */
	hipMalloc((void **)&dx, nbytes);
	if (dx == NULL) {
		printf("couldn't allocate GPU memory");
		return -1;
	}

	/* allocate CPU memory */
	hx = (float*) malloc(nbytes);
	if (hx == NULL) {
		printf("couldn't allocate CPU memory");
		return -2;
	}

	/* init */
	printf("hx original: \n");
	for (int i = 0; i < N; i++) {
		hx[i] = i;
		printf("%.1f", hx[i]);
	}

	/* copy data to GPU */
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

	/* call GPU */
	sum <<<1, N>>> (dx);

	/* let GPU finish */
	hipDeviceSynchronize();



	return 0;
}





