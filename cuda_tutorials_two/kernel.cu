#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define get_idx() (threadIdx.x)

__global__ void sum(float *x) {
	int idx = get_idx();
	x[idx] += 1;
}

int main() {
	int N = 32;
	int nbytes = N * sizeof(float);

	float *dx = NULL, *hx = NULL;

	/* allocate GPU memory */
	hipMalloc((void **)&dx, nbytes);
	if (dx == NULL) {
		printf("couldn't allocate GPU memory");
		return -1;
	}

	/* allocate CPU memory */
	hx = (float*) malloc(nbytes);
	//hipMalloc((void **)&hx, nbytes);

	if (hx == NULL) {
		printf("couldn't allocate CPU memory");
		return -2;
	}

	/* init */
	printf("hx original: \n");
	for (int i = 0; i < N; i++) {
		hx[i] = i;
		printf("%g\n", hx[i]);
	}

	/* copy data to GPU */
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

	/* call GPU */
	sum <<<1, N>>> (dx);

	/* let GPU finish */
	hipDeviceSynchronize();

	/* copy data from GPU */
	hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("%g\n", hx[i]);
	}
	
	return 0;
}





